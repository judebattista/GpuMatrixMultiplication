
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>

using namespace std;

// CUDA Kernel
//Performs matrix multiplication A * B = Out
//Note that aWidth must equal bHeight for the multiplication to succeed
//Thus we have summarily done away with the latter to remove temptation
//This kernel assumes that A is row major and B is column major
__global__ void matrixMultiply(double *matrixA, double *matrixB, double* matrixOut, 
                                int aHeight, int aWidth, int bWidth) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = row * bWidth + col;
 
    double sum = 0;
    // check to see if we are inside our problem space
    if (row < aHeight && col < bWidth) {
        // calculate row and col that we are going to compute
        // loop over A & B at the same time since A is row major and B is column major
        for (int ndx = 0; ndx < aWidth; ndx++) {
            double lhs = *(matrixA + row*aWidth + ndx); 
            double rhs = *(matrixB + col*aWidth + ndx);
            //Accumulate result
            sum += lhs * rhs; 
        }
        // store in matrix
        *(matrixOut + tid) = sum; 
    }
    
}

//CUDA Kernel using shared memory to speed things up.
//Performs matrix multiplication A * B = Out
//Note that aWidth must equal bHeight for the multiplication to succeed
//Thus we have summarily done away with the latter to remove temptation
//This kernel assumes that A is row major and B is column major
//Further the max (and probably optimal) aWidth value is 32.

//While the shared memory version does not currently work, due to issues with indexing into A and B
//the resultant calculation is still an order of magnitude faster than the naive implementation.
//How much of this is due to actual efficiency vs busted math (multiplying and adding zeroes instead of values), I am not sure.
//Averages over ten runs for each set of dimensions
// 128x128:     .0036 ms    vs. .0540 ms
// 256x256      .0035 ms    vs. .0590 ms 
// 1024x1024    .0044 ms    vs. .0880 ms
// 4096x4096    .0058 ms    vs. .0890 ms <-- I expected the naive kernel to take much longer on this set
__global__ void sharedMatrixMultiply(double *matrixA, double *matrixB, double* matrixOut, 
        int aHeight, int aWidth, int bWidth,
        double* sharedTestA, double* sharedTestB) {
    
    //Row and column of the output space
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = row * bWidth + col;
    //These values should correspond to our block size
    const int sharedWidth = 32;
    const int sharedHeight = 32;

    __shared__ double sharedA[sharedWidth * sharedHeight];
    __shared__ double sharedB[sharedWidth * sharedHeight];
   
    //figure out which rows of A and columns of B need to be loaded into the shared memory
    //This should be based off the TID for the output matrix
    //If we're in the first row of the output matrix, we need the first row of A
    //If we're in the first column of the output matrix, we need the first column of B
    //This correspondence seems to hold over the output space
    //The size of our block determines how many rows and columns we need to hold
    //For block 0,1 it needs to draw from the first set of rows and the second set of columns      
     
    //Each thread should load a single element from A and a single element from B into shared memory
    //Shared dimensions are NOT the same as block dimensions - should they be?
    //Let's assume they are - constraints make this reasonable
    int sharedCol = threadIdx.x;
    int sharedRow = threadIdx.y;
    *(sharedA + sharedRow * sharedWidth + sharedCol) = *(matrixA + row*aWidth + col); 
    *(sharedB + sharedRow * sharedWidth + sharedCol) = *(matrixB + row*aWidth + col);
    //Since the shared memory copy is not working, try something simpler
    //*(sharedA + sharedRow * sharedWidth + sharedCol) = blockIdx.x;
    //*(sharedB + sharedRow * sharedWidth + sharedCol) = blockIdx.y;
    __syncthreads();

    for(int ndx = 0; ndx < sharedHeight * sharedWidth; ndx++) {
        *(sharedTestA + ndx) = *(sharedA + ndx);
        *(sharedTestB + ndx) = *(sharedB + ndx);
    }

    double sum = 0;
    double lhs = 0;
    double rhs = 0;
    //TODO: CHECK YOUR SHARED MEMORY DIMENSIONS!
    // check to see if we are inside our problem space
    if (row < aHeight && col < bWidth) {
        // calculate row and col that we are going to compute
        // loop over A & B at the same time since A is row major and B is column major
        for (int ndx = 0; ndx < sharedWidth; ndx++) {
            lhs = *(sharedA + sharedRow*sharedWidth + ndx);
            rhs = *(sharedB + sharedCol*sharedWidth + ndx);
            //TODO: Test using the identity matrix as the RHS
            //rhs = 1;
            //Accumulate result
            sum += lhs * rhs; 
        }
        // store in matrix
        *(matrixOut + tid) = sum;
    }

}

void fillMatrix(double *target, int targetSize) {
    for (double ndx = 0; ndx < targetSize; ndx += 1) {
        *target = (int)ndx % 100;
        target++;
    }
}

void printMatrixRowMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << std::setw(7) << *(target + row * numCols + col) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

void printMatrixColMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << std::setw(7) << *(target + col * numRows + row) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

int main() {
    int aHeight = 4096;    //num of rows in A
    const int aWidth = 32;     //num of cols in A
    const int bHeight = 32;    //num of rows in B - this must be the same as aWidth for AB to work
    int bWidth = 4096;     //num of cols in B
    double *dev_matrixA, *dev_matrixB, *dev_matrixOut, *dev_sharedA, *dev_sharedB;
    hipEvent_t start, stop;
    float milliseconds; //how long did we take to do things?
    float naiveMs;
    float sharedMs;
    
    //bHeight = aWidth;   //Let's just make sure

    //allocate space
    double* matrixA = (double * )malloc(sizeof (double) * aHeight * aWidth);
    double* matrixB = (double * )malloc(sizeof (double) * bHeight * bWidth);        //The operand matrices
    double* matrixOut = (double * )malloc(sizeof (double) * aHeight * bWidth);      //The result matrix
    double* sharedA = (double * )malloc(sizeof (double) * 1024);      //The result matrix
    double* sharedB = (double * )malloc(sizeof (double) * 1024);      //The result matrix


    //fill operands
    fillMatrix(matrixA, aHeight * aWidth);
    fillMatrix(matrixB, bHeight * bWidth);

    //setup memory on device
    hipMalloc((void**)&dev_matrixA, (aHeight * aWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixB, (bHeight * bWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixOut, (aHeight * bWidth) * sizeof(double));
    hipMalloc((void**)&dev_sharedA, (1024) * sizeof(double));
    hipMalloc((void**)&dev_sharedB, (1024) * sizeof(double));

    // https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(dev_matrixA, matrixA, aHeight * aWidth * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrixB, matrixB, bHeight * bWidth * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrixOut, matrixOut, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice);

    //Set up problem space dimensions
    //dim3 threadsPerBlock (bWidth, aHeight);
    dim3 threadsPerBlock (32, 32);
    dim3 blocks (1, 4);
    //start timer event
    hipEventRecord(start);
    //call kernel
    matrixMultiply<<<blocks,threadsPerBlock>>>(dev_matrixA, dev_matrixB, dev_matrixOut, aHeight, aWidth, bWidth);
    //sharedMatrixMultiply<<<blocks,threadsPerBlock>>>(dev_matrixA, dev_matrixB, dev_matrixOut, aHeight, aWidth, bWidth, dev_sharedA, dev_sharedB);
    //stop timer event
    hipEventRecord(stop);

    //get result from device
    hipMemcpy(matrixOut, dev_matrixOut, aHeight * bWidth * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(matrixA, dev_matrixA, 16 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(matrixB, dev_matrixB,  16 * sizeof(double), hipMemcpyDeviceToHost);
     
    //calculate time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    naiveMs = milliseconds;

    //Test our calculation
    //printMatrixRowMaj(matrixA, aHeight, aWidth);
    //printMatrixColMaj(matrixB, bHeight, bWidth);
    //printMatrixRowMaj(matrixOut, aHeight, bWidth);
    //printMatrixRowMaj(sharedA, 2, 2);
    //printMatrixColMaj(sharedB, 2, 2);

    hipEventRecord(start);
    //call kernel
    //matrixMultiply<<<blocks,threadsPerBlock>>>(dev_matrixA, dev_matrixB, dev_matrixOut, aHeight, aWidth, bWidth);
    sharedMatrixMultiply<<<blocks,threadsPerBlock>>>(dev_matrixA, dev_matrixB, dev_matrixOut, aHeight, aWidth, bWidth, dev_sharedA, dev_sharedB);
    //stop timer event
    hipEventRecord(stop);

    //get result from device
    hipMemcpy(matrixOut, dev_matrixOut, aHeight * bWidth * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sharedA, dev_sharedA, 16 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sharedB, dev_sharedB,  16 * sizeof(double), hipMemcpyDeviceToHost);
     
    //calculate time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    sharedMs = milliseconds;

    //free memory
    hipFree(dev_matrixA);
    hipFree(dev_matrixB);
    hipFree(dev_matrixOut);
    hipFree(sharedA);
    hipFree(sharedB);

    std::cout << "the shared memory version took " << sharedMs << " milliseconds to complete.\n";
    std::cout << "the naive implementation took " << naiveMs << " milliseconds to complete.\n";

    return 0;
}
