
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;

// CUDA Kernel
//Performs matrix multiplication A * B = Out
//Note that aWidth must equal bHeight for the multiplication to succeed
//Thus we have summarily done away with the latter to remove temptation
__global__ void matrixMultiply(double *matrixA, double *matrixB, double* matrixOut, int aHeight, 
                                int aWidth, int bWidth) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = gridDim.x * gridDim.y * blockIdx.x * blockDim.x * blockDim.y + threadIdx.x;
    double sum = 0;
    // check to see if we are inside our problem space
    if (tid < aHeight * bWidth) {
        // calculate row and col that we are going to compute
        // loop over A & B at the same time since A is row major and B is column major
        for (int ndx = 0; ndx < aWidth; ndx++) {
            double lhs = *(matrixA + row*aWidth + ndx);
            double rhs = *(matrixB + col*aWidth + ndx);
            //Accumulate result
            sum += lhs * rhs; 
        }
        // store in matrix
        *(matrixOut + tid) = sum;
    }
    
}

void fillMatrix(double *target, int targetSize) {
    for (double ndx = 0; ndx < targetSize; ndx += 1) {
        *target = ndx;
        target++;
    }
}

void printMatrixRowMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << *(target + row * numCols + col) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

void printMatrixColMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << *(target + col * numRows + row) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

int main() {
    int aHeight = 3;    //num of rows in A
    int aWidth = 2;     //num of cols in A
    int bHeight = 2;    //num of rows in B - this must be the same as aWidth for AB to work
    int bWidth = 3;     //num of cols in B
    double *dev_matrixA, *dev_matrixB, *dev_matrixOut;
    hipEvent_t start, stop;
    float milliseconds; //how long did we take to do things?

    bHeight = aWidth;   //Let's just make sure

    //allocate space
    double* matrixA = (double * )malloc(sizeof (double) * aHeight * aWidth);
    double* matrixB = (double * )malloc(sizeof (double) * bHeight * bWidth);        //The operand matrices
    double* matrixOut = (double * )malloc(sizeof (double) * aHeight * bWidth);      //The result matrix

    //fill operands
    fillMatrix(matrixA, aHeight * aWidth);
    fillMatrix(matrixB, bHeight * bWidth);

    //setup memory shit
    hipMalloc((void**)&dev_matrixA, (aHeight * aWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixB, (bHeight * bWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixOut, (aHeight * bWidth) * sizeof(double));

    // https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(dev_matrixA, matrixA, aHeight * aWidth * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrixB, matrixB, bHeight * bWidth * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrixOut, matrixOut, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice);

    //start timer event
    hipEventRecord(start);
    //call kernel
    dim3 threadsPerBlock (1, 32);
    dim3 blocks (32, 32);
    matrixMultiply<<<1,threadsPerBlock>>>(dev_matrixA, dev_matrixB, dev_matrixOut, aHeight, aWidth, bWidth);
    //stop timer event
    hipEventRecord(stop);

    //get result
    hipMemcpy(matrixOut, dev_matrixOut, aHeight * bWidth * sizeof(double), hipMemcpyDeviceToHost);
    
    //calculate time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    //free memory
    hipFree(dev_matrixA);
    hipFree(dev_matrixB);
    hipFree(dev_matrixOut);

    printMatrixRowMaj(matrixA, aHeight, aWidth);
    printMatrixColMaj(matrixB, bHeight, bWidth);
    printMatrixRowMaj(matrixOut, aHeight, bWidth);



    return 0;
}