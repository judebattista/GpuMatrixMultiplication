
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>

using namespace std;

// CUDA Kernel
//Performs matrix multiplication A * B = Out
//Note that aWidth must equal bHeight for the multiplication to succeed
//Thus we have summarily done away with the latter to remove temptation
//This kernel assumes that A is row major and B is column major
__global__ void matrixMultiply(double *matrixA, double *matrixB, double* matrixOut, 
                                int aHeight, int aWidth, int bWidth) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = row * bWidth + col;
 
    double sum = 0;
    // check to see if we are inside our problem space
    if (row < aHeight && col < bWidth) {
        // calculate row and col that we are going to compute
        // loop over A & B at the same time since A is row major and B is column major
        for (int ndx = 0; ndx < aWidth; ndx++) {
            double lhs = *(matrixA + row*aWidth + ndx); 
            double rhs = *(matrixB + col*aWidth + ndx);
            //Accumulate result
            sum += lhs * rhs; 
        }
        // store in matrix
        *(matrixOut + tid) = sum; 
    }
    
}

__global__ void sharedMatrixMultiply(double *matrixA, double *matrixB, double* matrixOut, 
        int aHeight, int aWidth, int bWidth,
        double* sharedTestA, double* sharedTestB) {
    
    //Row and column of the output space
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = row * bWidth + col;
    const int sharedWidth = 2;
    const int sharedHeight = 2;

    __shared__ double sharedA[sharedWidth * sharedHeight];
    __shared__ double sharedB[sharedWidth * sharedHeight];
   
    //figure out which rows of A and columns of B need to be loaded into the shared memory
    //This should be based off the TID for the output matrix
    //If we're in the first row of the output matrix, we need the first row of A
    //If we're in the first column of the output matrix, we need the first column of B
    //This correspondence seems to hold over the output space
    //The size of our block determines how many rows and columns we need to hold
    //For block 0,1 it needs to draw from the first set of rows and the second set of columns      
     
    //Each thread should load a single element from A and a single element from B into shared memory
    //The index into A/B is found by:
    //Shared dimensions are NOT the same as block dimensions
    int sharedCol = threadIdx.x;
    int sharedRow = threadIdx.y;
    *(sharedA + sharedRow * sharedWidth + sharedCol) = *(matrixA + row*aWidth + col); 
    *(sharedB + sharedRow * sharedWidth + sharedCol) = *(matrixB + row*aWidth + col);
    __syncthreads();

    for(int ndx = 0; ndx < sharedHeight * sharedWidth; ndx++) {
        *(sharedTestA + ndx) = *(sharedA + ndx);
        *(sharedTestB + ndx) = *(sharedB + ndx);
    }

    double sum = 0;
    double lhs = 0;
    double rhs = 0;
    //TODO: CHECK YOUR SHARED MEMORY DIMENSIONS!
    // check to see if we are inside our problem space
    if (row < aHeight && col < bWidth) {
        // calculate row and col that we are going to compute
        // loop over A & B at the same time since A is row major and B is column major
        for (int ndx = 0; ndx < sharedWidth; ndx++) {
            lhs = *(sharedA + sharedRow*sharedWidth + ndx);
            //rhs = *(sharedB + sharedCol*sharedWidth + ndx);
            //TODO: Using the identity matrix as the RHS
            rhs = 1;
            //Accumulate result
            sum += lhs * rhs; 
        }
        // store in matrix
        *(matrixOut + tid) = sum;
    }

}

void fillMatrix(double *target, int targetSize) {
    for (double ndx = 0; ndx < targetSize; ndx += 1) {
        *target = ndx;
        target++;
    }
}

void printMatrixRowMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << std::setw(7) << *(target + row * numCols + col) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

void printMatrixColMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << std::setw(7) << *(target + col * numRows + row) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

int main() {
    int aHeight = 8;    //num of rows in A
    const int aWidth = 4;     //num of cols in A
    const int bHeight = 4;    //num of rows in B - this must be the same as aWidth for AB to work
    int bWidth = 8;     //num of cols in B
    double *dev_matrixA, *dev_matrixB, *dev_matrixOut, *dev_sharedA, *dev_sharedB;
    hipEvent_t start, stop;
    float milliseconds; //how long did we take to do things?

    //bHeight = aWidth;   //Let's just make sure

    //allocate space
    double* matrixA = (double * )malloc(sizeof (double) * aHeight * aWidth);
    double* matrixB = (double * )malloc(sizeof (double) * bHeight * bWidth);        //The operand matrices
    double* matrixOut = (double * )malloc(sizeof (double) * aHeight * bWidth);      //The result matrix
    double* sharedA = (double * )malloc(sizeof (double) * 1024);      //The result matrix
    double* sharedB = (double * )malloc(sizeof (double) * 1024);      //The result matrix


    //fill operands
    fillMatrix(matrixA, aHeight * aWidth);
    fillMatrix(matrixB, bHeight * bWidth);

    //setup memory on device
    hipMalloc((void**)&dev_matrixA, (aHeight * aWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixB, (bHeight * bWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixOut, (aHeight * bWidth) * sizeof(double));
    hipMalloc((void**)&dev_sharedA, (1024) * sizeof(double));
    hipMalloc((void**)&dev_sharedB, (1024) * sizeof(double));

    // https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMemcpy(dev_matrixA, matrixA, aHeight * aWidth * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrixB, matrixB, bHeight * bWidth * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_matrixOut, matrixOut, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice);

    //Set up problem space dimensions
    //dim3 threadsPerBlock (bWidth, aHeight);
    dim3 threadsPerBlock (4, 4);
    dim3 blocks (8, 8);
    //start timer event
    hipEventRecord(start);
    //call kernel
    //matrixMultiply<<<blocks,threadsPerBlock>>>(dev_matrixA, dev_matrixB, dev_matrixOut, aHeight, aWidth, bWidth);
    sharedMatrixMultiply<<<blocks,threadsPerBlock>>>(dev_matrixA, dev_matrixB, dev_matrixOut, aHeight, aWidth, bWidth, dev_sharedA, dev_sharedB);
    //stop timer event
    hipEventRecord(stop);

    //get result from device
    hipMemcpy(matrixOut, dev_matrixOut, aHeight * bWidth * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sharedA, dev_sharedA, 16 * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(sharedB, dev_sharedB,  16 * sizeof(double), hipMemcpyDeviceToHost);
     
    //calculate time
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    //free memory
    hipFree(dev_matrixA);
    hipFree(dev_matrixB);
    hipFree(dev_matrixOut);
    hipFree(sharedA);
    hipFree(sharedB);

    //Test our calculation
    printMatrixRowMaj(matrixA, aHeight, aWidth);
    printMatrixColMaj(matrixB, bHeight, bWidth);
    printMatrixRowMaj(matrixOut, aHeight, bWidth);
    printMatrixRowMaj(sharedA, 2, 2);
    printMatrixColMaj(sharedB, 2, 2);



    return 0;
}
