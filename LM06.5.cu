
#include <hip/hip_runtime.h>
#include <iomanip>
#include <iostream>

using namespace std;

// CUDA Kernel
//Performs matrix multiplication A * B = Out
//Note that aWidth must equal bHeight for the multiplication to succeed
//Thus we have summarily done away with the latter to remove temptation
//This kernel assumes that A is row major and B is column major
__global__ void matrixMultiply(double *matrixA, double *matrixB, double* matrixOut, 
                                int aHeight, int aWidth, int bWidth) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = row * bWidth + col;
 
    double sum = 0;
    // check to see if we are inside our problem space
    if (row < aHeight && col < bWidth) {
        // calculate row and col that we are going to compute
        // loop over A & B at the same time since A is row major and B is column major
        for (int ndx = 0; ndx < aWidth; ndx++) {
            double lhs = *(matrixA + row*aWidth + ndx); 
            double rhs = *(matrixB + col*aWidth + ndx);
            //Accumulate result
            sum += lhs * rhs; 
        }
        // store in matrix
        *(matrixOut + tid) = sum; 
    }
    
}

// CUDA Kernel
//Performs matrix multiplication A + B = Out
//Both operand matrices must be square and have the same dimension
__global__ void matrixAdd(double *matrixA, double *matrixB, double* matrixOut, 
                                int aHeight, int aWidth) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int tid = row * aWidth + col;
 
    // check to see if we are inside our problem space
    if (row < aHeight && col < aWidth) {
        *(matrixOut + tid) = *(matrixA + tid) + *(matrixB + tid);    
    }   
}

void fillMatrix(double *target, int targetSize) {
    for (double ndx = 0; ndx < targetSize; ndx += 1) {
        *target = (int)ndx % 100;
        target++;
    }
}

void printMatrixRowMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << std::setw(7) << *(target + row * numCols + col) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

void printMatrixColMaj(double *target, int numRows, int numCols) {
    for (int row = 0; row < numRows; row++) {
        for (int col = 0; col < numCols; col++) {
            std::cout << std::setw(7) << *(target + col * numRows + row) << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std:: endl;
}

int main() {
    int aHeight = 32;    //num of rows in A
    const int aWidth = 32;     //num of cols in A
    const int bHeight = 32;    //num of rows in B - this must be the same as aWidth for AB to work
    int bWidth = 32;     //num of cols in B
    //For simplicity's sake we will assume that C has the same dimensions as C and D has the same dimensions as B
    double *dev_matrixA, *dev_matrixB, *dev_matrixProd1;
    double *dev_matrixC, *dev_matrixD, *dev_matrixProd2;
    double *dev_matrixSum;
    hipStream_t stream1, stream2;
    hipEvent_t start, stop, mult1done, mult2done;
    float milliseconds; //how long did we take to do things?
    
    //bHeight = aWidth;   //Let's just make sure

    //allocate space
    double* matrixA = (double * )malloc(sizeof (double) * aHeight * aWidth);
    double* matrixB = (double * )malloc(sizeof (double) * bHeight * bWidth);        //The operand matrices for the first mult
    double* matrixC = (double * )malloc(sizeof (double) * aHeight * aWidth);
    double* matrixD = (double * )malloc(sizeof (double) * bHeight * bWidth);        //The operand matrices for the second mult

    double* matrixProd1 = (double * )malloc(sizeof (double) * aHeight * bWidth);      //The result matrix of the first mult
    double* matrixProd2 = (double * )malloc(sizeof (double) * aHeight * bWidth);      //The result matrix of the second mult
    double* matrixSum = (double * )malloc(sizeof (double) * aHeight * bWidth);      //The result matrix

    //fill operands
    fillMatrix(matrixA, aHeight * aWidth);
    fillMatrix(matrixB, bHeight * bWidth);
    fillMatrix(matrixC, aHeight * aWidth);
    fillMatrix(matrixD, bHeight * bWidth);

    //setup memory on device
    hipMalloc((void**)&dev_matrixA, (aHeight * aWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixB, (bHeight * bWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixProd1, (aHeight * bWidth) * sizeof(double));

    hipMalloc((void**)&dev_matrixC, (aHeight * aWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixD, (bHeight * bWidth) * sizeof(double));
    hipMalloc((void**)&dev_matrixProd2, (aHeight * bWidth) * sizeof(double));
    
    hipMalloc((void**)&dev_matrixSum, (aHeight * bWidth) * sizeof(double));
    

    //Set up problem space dimensions
    //dim3 threadsPerBlock (bWidth, aHeight);
    dim3 threadsPerBlock (32, 32);
    dim3 blocks (1,1);

    //Create streams
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventCreate(&mult1done);
    hipEventCreate(&mult2done);
   
    hipEventRecord(start); 

    //Load the operands for the first multiplication
    hipMemcpyAsync(dev_matrixA, matrixA, aHeight * aWidth * sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_matrixB, matrixB, bHeight * bWidth * sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_matrixProd1, matrixProd1, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice, stream1);

    //call multiply kernel in stream 1
    matrixMultiply<<<blocks,threadsPerBlock, 0, stream1>>>(dev_matrixA, dev_matrixB, dev_matrixProd1, aHeight, aWidth, bWidth);
    
    //While the first multiply is running, load the operands for the second multiplication
    hipMemcpyAsync(dev_matrixC, matrixC, aHeight * aWidth * sizeof(double), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(dev_matrixD, matrixD, bHeight * bWidth * sizeof(double), hipMemcpyHostToDevice, stream2);
    hipMemcpyAsync(dev_matrixProd2, matrixProd2, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice, stream2);

    //call multiply kernel in stream 2
    matrixMultiply<<<blocks,threadsPerBlock, 0, stream2>>>(dev_matrixC, dev_matrixD, dev_matrixProd2, aHeight, aWidth, bWidth);

    //get result from device
    hipMemcpyAsync(matrixProd1, dev_matrixProd1, aHeight * bWidth * sizeof(double), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(matrixProd2, dev_matrixProd2, aHeight * bWidth * sizeof(double), hipMemcpyDeviceToHost, stream2);
    
    //Ensure that both stream1 and stream2 are done
    //cudaEventRecord(mult1done, stream1); //<-- If we use stream1 to perform the addition, we do not need this event. Stream1 won't continue until it's finished its memcpy
    hipEventRecord(mult2done, stream2);

    //Make sure the second multiplication is done before continuing
    hipStreamWaitEvent(stream1, mult2done, 0);

    //Copy the multiplication results to the device 
    hipMemcpyAsync(dev_matrixProd1, matrixProd1, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_matrixProd2, matrixProd2, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(dev_matrixSum, matrixSum, aHeight * bWidth * sizeof(double), hipMemcpyHostToDevice, stream1);
    
    //Call the addition kernel
    matrixAdd<<<blocks,threadsPerBlock, 0, stream1>>>(dev_matrixProd1, dev_matrixProd2, dev_matrixSum, aHeight, aWidth);

    //Get the result
    hipMemcpyAsync(matrixSum, dev_matrixSum, aHeight * bWidth * sizeof(double), hipMemcpyDeviceToHost, stream1);
    
    //calculate time
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    
    //Test our calculation
    printMatrixRowMaj(matrixA, aHeight, aWidth);
    printMatrixColMaj(matrixB, bHeight, bWidth);
    printMatrixRowMaj(matrixProd1, aHeight, bWidth);
    printMatrixRowMaj(matrixProd2, aHeight, bWidth);
    printMatrixRowMaj(matrixSum, aHeight, bWidth);

    //free memory
    hipFree(dev_matrixA);
    hipFree(dev_matrixB);
    hipFree(dev_matrixProd1);
    hipFree(dev_matrixC);
    hipFree(dev_matrixB);
    hipFree(dev_matrixProd1);
    hipFree(dev_matrixSum);

    std::cout << "It took " << milliseconds << " milliseconds to complete.\n";

    return 0;
}
